#include "hip/hip_runtime.h"
/*
  Copyright 2009-2019 Lianqi Wang <lianqiw-at-tmt-dot-org>
  
  This file is part of Multithreaded Adaptive Optics Simulator (MAOS).

  MAOS is free software: you can redistribute it and/or modify it under the
  terms of the GNU General Public License as published by the Free Software
  Foundation, either version 3 of the License, or (at your option) any later
  version.

  MAOS is distributed in the hope that it will be useful, but WITHOUT ANY
  WARRANTY; without even the implied warranty of MERCHANTABILITY or FITNESS FOR
  A PARTICULAR PURPOSE.  See the GNU General Public License for more details.

  You should have received a copy of the GNU General Public License along with
  MAOS.  If not, see <http://www.gnu.org/licenses/>.
*/

#include "curmat.h"
#include "cucmat.h"
#include "utils.h"
#include "kernel.h"

void curset(curmat &A, Real alpha, hipStream_t stream){
    if(A && A()){
	set_do<<<DIM(A.Nx()*A.Ny(),256),0,stream>>>(A(), alpha, A.Nx()*A.Ny());
    }
}

void curcp(curmat &out, const curmat &in, hipStream_t stream){
    if(!in){
	Zero(out, stream);
    }else{
	if(!out){
	    out=curmat(in.Nx(), in.Ny());
	}else{
	    assert(out.N() == in.N());
	}
	hipMemcpyAsync(out(), in(), in.N()*sizeof(Real), MEMCPY_D2D, stream);
    }
}
void curcp(curmat &out, const curmat &in){
    if(!in){
	Zero(out);
    }else{
	if(!out){
	    out=curmat(in.Nx(), in.Ny());
	}else{
	    assert(out.N() == in.N());
	}
	hipMemcpy(out(), in(), in.N()*sizeof(Real), MEMCPY_D2D);
    }
}

/**
   out=out*beta+in*alpha;
*/
void curadd(curmat &out, Real alpha, const curmat &in, Real beta, hipStream_t stream){
    if(!in) return;
    if(!out || alpha==0){
	curcp(out, in, stream);
	if(Z(fabs)(beta-(Real)1)>EPS){
	    scale_do<<<DIM(in.Nx()*in.Ny(), 256),0,stream>>>
		(out(), in.Nx()*in.Ny(), beta);
	}
    }else{
	assert(in.N()==out.N());
	add_do<<<DIM(in.Nx()*in.Ny(), 256),0,stream>>>
	    (out(), NULL, alpha, in(), NULL, beta, in.Nx()*in.Ny());
    }
}
/**
   out=out*beta+abs2(in)*alpha;
*/
void curaddcabs2(curmat &out, Real alpha, const cucmat &in, Real beta, hipStream_t stream){
    if(!out){
	out=curmat(in.Nx(),in.Ny());
    }else if(alpha==0){
	Zero(out, stream);
    }
    addcabs2_do<<<DIM(in.Nx()*in.Ny(), 256),0,stream>>>
	(out(), alpha, in(), beta, in.Nx()*in.Ny());
}
void curscale(curmat &in, Real alpha, hipStream_t stream){
    if(!in) return;
    if(alpha==0) {
	Zero(in, stream);
    }else if(Z(fabs)(alpha-(Real)1)>EPS){
	int n=in.Nx()*in.Ny();
	scale_do<<<DIM(n,256), 0, stream>>>(in(), n, alpha); 
    }
}

/**
   Computes C = alpha * C + beta * op(A) * B ;
*/
void curmm(curmat &C, Real alpha, const curmat &A, const curmat &B, const char trans[2], Real beta, stream_t &stream){
    int m,n,k,k2;
    hipblasOperation_t transa, transb;
    if(trans[0]=='t'){
	m=A.Ny();
	k=A.Nx();
	transa=HIPBLAS_OP_T;
    }else{
	m=A.Nx();
	k=A.Ny();
	transa=HIPBLAS_OP_N;
    }
    if(trans[1]=='t'){
	n=B.Nx();
	k2=B.Ny();
	transb=HIPBLAS_OP_T;
    }else{
	n=B.Ny();
	k2=B.Nx();
	transb=HIPBLAS_OP_N;
    }
    if(!C){
	C=curmat(m,n);
    }else{
	assert(C.Nx()==m && C.Ny()==n);
    }
    if(k!=k2) error("Matrix mismatch\n");
    DO(CUBL(gemm)(stream.blas(), transa, transb, m,n,k,
		  &beta, A(), A.Nx(), B(), B.Nx(), &alpha, C(), C.Nx()));
}
/**
   Computes C = alpha * C + beta * op(A) * B ;
*/
void curmv(Real *c, Real alpha, const curmat &A, const Real *b, char trans, Real beta, stream_t &stream){
    CUBL(gemv)(stream.blas(), (trans=='t'||trans==1)?HIPBLAS_OP_T:HIPBLAS_OP_N, A.Nx(), A.Ny(), &beta, A(), A.Nx(), b, 1, &alpha, c, 1);
}
void curcellmm(curcell &C, Real alpha, const curcell &A, const curcell &B, 
	       const char trans[2], const double beta, stream_t &stream){
    if(!A || !B) return;
    int ax, az;
    int nx,ny,nz;
    int bz, by;
    if(trans[0]=='n'||trans[0]=='N'){
	nx=A.Nx(); 
	ax=1; az=A.Nx();
	nz=A.Ny();
    }else{ 
	nx=A.Ny();
	az=1; ax=A.Nx();
	nz=A.Nx();
    }
    if(trans[1]=='n'||trans[0]=='N'){
	ny=B.Ny(); 
	bz=1; by=B.Nx();
	if(nz!=B.Nx()) error("miX(mat)ch\n");
    }else{
	ny=B.Nx();
	by=1; bz=B.Nx();
	if(nz!=B.Ny()) error("miX(mat)ch\n");
    }
    if(!C){
	C=curcell(nx,ny);
    }else{
	assert(C.Nx()==nx && C.Ny()==ny);
	if(alpha==0){
	    Zero(C, stream);
	}else if(Z(fabs)(alpha-(Real)1)>EPS){
	    curcellscale(C, alpha, stream);
	}
    }
    for(int iy=0; iy<ny; iy++){
	for(int ix=0; ix<nx; ix++){
	    for(int iz=0; iz<nz; iz++){
		if(A[ix*ax+iz*az]&&B[iz*bz+iy*by]){
		    curmm(C[ix+iy*nx],1.,A[ix*ax+iz*az], 
			  B[iz*bz+iy*by],trans,beta,stream);
		}
	    }
	}
    }
}

/*
  A=A*beta+B*alpha;
*/
void curcelladd(curcell &A, Real beta, const curcell &B, Real alpha, hipStream_t stream){
    if(!B) return;
    if(!A){
	A=New(B);
    }else{
	assert(A.Nx()==B.Nx() && A.Ny()==B.Ny());
    }
    if(A.M() && B.M()){
	curadd(A.M(), beta, B.M(), alpha, stream);
    }else{
	for(int i=0; i<B.Nx()*B.Ny(); i++){
	    curadd(A[i], beta, B[i], alpha,stream);
	}
    }
}

void curadd(curmat &A, Real beta, hipStream_t stream){
    const int n=A.Nx()*A.Ny();
    add_do<<<DIM(n, 256), 0, stream>>>(A(), beta, n);
}
/**
   add a vector to another, scaled by alpha and beta. all in device memory.
   a=a+b*alpha*beta;
*/

/**
   out=out+in*alpha; beta, alpha lives in device memory.
*/
void curadd(curmat &out, const curmat &in, Real *alpha, Real alpha2, hipStream_t stream){
    if(!out){
	out=curmat(in.Nx(), in.Ny());
    }
    add_do<<<DIM(in.Nx()*in.Ny(), 256),0,stream>>>
	(out(), in(), alpha, alpha2, in.Nx()*in.Ny());
}


/**
   A=A*beta+B*alpha; beta, alpha lives in device memory.
*/
void curcelladd(curcell &A, const curcell &B, Real* alpha, Real alpha2, hipStream_t stream){
    if(!B) return;
    if(!A){
	A=New(B);
    }else{
	assert(A.Nx()==B.Nx() && A.Ny()==B.Ny());
    }
    if(A.M() && B.M()){
	curadd(A.M(), B.M(), alpha, alpha2, stream);
    }else{
	for(int i=0; i<B.N(); i++){
	    curadd(A[i], B[i], alpha, alpha2,  stream);
	}
    }
}

/**
   out=out*beta+in; beta, alpha lives in device memory.
*/
void curadd(curmat &out, Real *alpha1, const curmat &in, hipStream_t stream){
    if(!out){
	out=curmat(in.Nx(), in.Ny());
    }
    add_do<<<DIM(in.Nx()*in.Ny(), 256),0,stream>>>
	(out(), alpha1, 1.f, in(), in.Nx()*in.Ny());
}

/**
   A=A*alpha1+B*alpha; alpha1, alpha lives in device memory.
*/
void curcelladd(curcell &A, Real* alpha1, const curcell &B, hipStream_t stream){
    if(!B) return;
    if(!A){
	A=New(B);
    }else{
	assert(A.Nx()==B.Nx() && A.Ny()==B.Ny());
    }
    if(A.M() && B.M()){
	curadd(A.M(), alpha1, B.M(), stream);
    }else{
	for(int i=0; i<B.Nx()*B.Ny(); i++){
	    curadd(A[i], alpha1, B[i],  stream);
	}
    }
}



Real curinn(const curmat &a, const curmat &b, hipStream_t stream){
    curmat res(1,1);
    Real out;
    inn_wrap(res(), a(), b(), a.Nx()*a.Ny(), stream);
    CUDA_SYNC_STREAM;
    hipMemcpy(&out, res(), sizeof(Real), hipMemcpyDeviceToHost);
    return out;
}

/**
   Sum all the elements in an array.
 */
void cursum2(Real *restrict res,/**<Result in GPU*/
	     const curmat &a,   /**<Source in GPU*/
	     hipStream_t stream){
    hipMemsetAsync(res, 0, sizeof(Real), stream);
    sum_wrap(res, a(), a.Nx()*a.Ny(), stream);
}
/**
   Sum all the elements in an array, and return a value.
*/
Real cursum(const curmat &a, hipStream_t stream){
    Real out;//result in CPU.
    curmat res(1,1);
    sum_wrap(res, a(), a.Nx()*a.Ny(), stream);
    CUDA_SYNC_STREAM;
    hipMemcpy(&out, res(), sizeof(Real), hipMemcpyDeviceToHost);
    return out;
}

/**
   Find the maximum value
*/
Real curmax(const curmat &a, hipStream_t stream){
    Real out;
    curmat res(1,1);
    max_wrap(res, a(), a.N(), stream);
    CUDA_SYNC_STREAM;
    hipMemcpy(&out, res(), sizeof(Real), hipMemcpyDeviceToHost);
    hipFree(res);
    return out;
}

/**
   Find the maximum value
*/
Real curmaxabs(const curmat &a, hipStream_t stream){
    Real out;
    curmat res(1,1);
    maxabs_wrap(res, a(), a.N(), stream);
    CUDA_SYNC_STREAM;
    hipMemcpy(&out, res(), sizeof(Real), hipMemcpyDeviceToHost);
    hipFree(res);
    return out;
}
/**
   Find the maximum value
*/
Real curcellmax(const curcell &a, hipStream_t stream){
    int n=a.Nx()*a.Ny();
    Real out;
    curmat res(n+1,1);
    for(int i=0; i<n; i++){
	int m=a[i].N();
	max_wrap(&res[i], a[i](), m, stream);
    }
    if(n>1) {
	max_wrap(&res[n], res, n, stream);
    }
    CUDA_SYNC_STREAM;
    hipMemcpy(&out, &res[n>1?n:0], sizeof(Real), hipMemcpyDeviceToHost);
    return out;
}
/**
   Find the maximum value
*/
Real curcellmaxabs(const curcell &a, hipStream_t stream){
    int n=a.N();
    Real out;
    curmat res(n+1,1);
    for(int i=0; i<n; i++){
	int m=a[i].N();
	maxabs_wrap(&res[i], a[i](), m, stream);
    }
    if(n>1) {
	maxabs_wrap(&res[n], res, n, stream);
    }
    CUDA_SYNC_STREAM;
    hipMemcpy(&out, &res[n>1?n:0], sizeof(Real), hipMemcpyDeviceToHost);
    return out;
}
/**
   Scale elements
*/
void curcellscale(curcell &A, Real alpha, hipStream_t stream){
    if(!A) return;
    if(A.M()){
	curscale(A.M(), alpha, stream);
    }else{
	for(int i=0; i<A.Nx()*A.Ny(); i++){
	    curscale(A[i], alpha, stream);
	}
    }
}
