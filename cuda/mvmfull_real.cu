#include "hip/hip_runtime.h"
/*
  Copyright 2009-2019 Lianqi Wang <lianqiw-at-tmt-dot-org>
  
  This file is part of Multithreaded Adaptive Optics Simulator (MAOS).

  MAOS is free software: you can redistribute it and/or modify it under the
  terms of the GNU General Public License as published by the Free Software
  Foundation, either version 3 of the License, or (at your option) any later
  version.

  MAOS is distributed in the hope that it will be useful, but WITHOUT ANY
  WARRANTY; without even the implied warranty of MERCHANTABILITY or FITNESS FOR
  A PARTICULAR PURPOSE.  See the GNU General Public License for more details.

  You should have received a copy of the GNU General Public License along with
  MAOS.  If not, see <http://www.gnu.org/licenses/>.
*/
#include <errno.h>
#include "utils.h"
#include "curmat.h"
#include "cucmat.h"
/**
\file mvmfull_real.cu

2013-11-12

Involved from mvmfull_iwfs.cu. Take into account real WFS geometry.
Accuracy Verified.
Test MVM for a single WFS, with networking. ethtest is the server.

two ways: 
1) broadcast gradients to both GPUs with each GPU handle part of actuators. (not tested yet)
2) partition gradients to GPUs with GPU handle all actuators. (selected)

Use schedtool -a 0x1 PID to let the exe only run one CPU 0. This prevents jitter.

For GTX590, use NOVER=2 and NGRID=30 gives best performance.
*/

static unsigned int event_flag=hipEventDisableTiming;
typedef struct{
    curmat cumvm;//active mvm control matrix
    curmat cumvm_next;//inactive mvm control matrix.
    curmat cumvm1;
    curmat cumvm2;
    curcell mtch;
    Array<short,Gpu> pix;//pixels. Each sa has 15x6=90 pixels.
    Array<short,Gpu> pixbias;
    curmat grad;
    curmat gdm;/*add to grad*/
    curmat act;
    curcell actelse;
    Real FSMdelta; /*FSM actual angle difference from command.*/
    curcell im0;
    int mtch_isa;
    cuimat saind;
    stream_t stream_p;//pixels and other transportation across PCI-E.
    stream_t stream_g;//grads
    Array<stream_t> stream_a;//act
    stream_t stream_b;//background process
    int ism;//index of stream for mvm
    int count;
    int gpu;//Which GPU this data is for
    int istep;//Which time step we are in
    int copy_mvm;//1: need to copy mvm.
    int ic;//the column that we are copying.
    hipEvent_t *event_p;
    hipEvent_t *event_g;
    hipEvent_t event_pall;
    event_t *event_w;
}GPU_DATA_T;
/*Does matched filter
  threadIdx.x is 32, for 1/3 of pixels
  threadIdx.y is for a few subapertures
  no need to sync threads as every 32 is in the same warp.
*/
static void __global__ mtch_do(const Real *mtch, const short *pix, const short *pixbias, Real *grad,
			       int *saind, int nsa){
    extern __shared__ Real cum[];//for cumulation and reduction
    for(int isa=threadIdx.y+blockDim.y*blockIdx.x; isa<nsa; isa+=blockDim.y*gridDim.x){
	const int npix=saind[isa+1]-saind[isa];
	Real *cumx=cum+threadIdx.y*blockDim.x*2;
	Real *cumy=cumx+blockDim.x;
	const short *pixi=pix+saind[isa];
	const short *pixbiasi=pixbias+saind[isa];
	const Real *mtchx=mtch+saind[isa]*2;
	const Real *mtchy=mtchx+npix;
	const int npix3=npix/3;
	//sum 3 times for max 90 pixels.
	int ipix=threadIdx.x;
	cumx[ipix]=0;
	cumy[ipix]=0;
	if(ipix<npix3){
	    cumx[ipix]=mtchx[ipix]*(pixi[ipix]-pixbiasi[ipix])
		+mtchx[ipix+npix3]*(pixi[ipix+npix3]-pixbiasi[ipix+npix3])
		+mtchx[ipix+npix3*2]*(pixi[ipix+npix3*2]-pixbiasi[ipix+npix3*2]);
	    cumy[ipix]=mtchy[ipix]*(pixi[ipix]-pixbiasi[ipix])
		+mtchy[ipix+npix3]*(pixi[ipix+npix3]-pixbiasi[ipix+npix3])
		+mtchy[ipix+npix3*2]*(pixi[ipix+npix3*2]-pixbiasi[ipix+npix3*2]);
	}
	//reduction
	for(int step=16;step>0;step>>=1){
	    if(ipix<step){
		cumx[ipix]+=cumx[ipix+step];
		cumy[ipix]+=cumy[ipix+step];
	    }
	}
	if(ipix==0){
	    grad[isa*2]=cumx[0];
	    grad[isa*2+1]=cumy[0];
	}
    }
}
/*
  Accumulate statistics
*/
static void __global__ dither_acc_do(const short *pix,
				     Real *im0, Real *imx, Real *imy, 
				     Real cd, Real sd, int totpix){
    for(int ipix=threadIdx.x+blockIdx.x*blockDim.x; ipix<totpix; ipix+=blockDim.x*gridDim.x){
	short ii=pix[ipix];//-pixbias[ipix];
	im0[ipix]+=ii;
	imx[ipix]+=ii*cd;
	imy[ipix]+=ii*sd;
    }
}

    // First calibrate out pixels
    // im0/=imc; imx*=(2/a2m*imc); imy*=2/(a2m*imc)
/**
   A standalone routine that testes applying MVM for a single WFS and update mvm.
   The orderig of gradients are like xyxyxy instead of normal xxxyyy.

   Important: 
   1) Only page locked host memory can do async memcpy that overallps with computation
   2) Has to be Portable for multiple GPUs to do async memcpy concurrently.
   
*/
void mvmfull_real(int *gpus, int ngpu, int nstep){
    dbg("Using %d gpus. nstep=%d\n", ngpu, nstep);
    int nstep0=nstep>1?1000:0;//for warm up
    //Load subaperture actual pixel numbers along radial direction and offset of pixel of each subaperture.
    dmat *d_saind=dread("NFIRAOS_saind");
    const int nsa=d_saind->nx-1;
    int *saind=(int*)malloc(sizeof(int)*(1+nsa));
    for(int i=0; i<nsa+1; i++){
	saind[i]=(int)d_saind->p[i];
    }
    dfree(d_saind);
    const int totpix=saind[nsa];
    const int nact=6981;//active subapertures.
    int ng=nsa*2;
    X(mat) *mvm1, *mvm2, *pix1, *pix2, *mtch, *ptt, *pixbias;
    X(mat) *im0;
    if(zfexist("mvm2.bin")){
	mvm1=X(read)("mvm1");
	mvm2=X(read)("mvm2");
	pix1=X(read)("pix1");
	pix2=X(read)("pix2");
	mtch=X(read)("mtch");
	ptt=X(read)("ptt");
	pixbias=X(read)("pixbias");
    }else{
	mvm1=X(new)(nact, ng);
	mvm2=X(new)(nact, ng);
	pix1=X(new)(totpix,1);
	pix2=X(new)(totpix,1);
	mtch=X(new)(totpix*2,1);
	ptt=X(new)(ng, 2);
	pixbias=X(new)(totpix, 1);
	rand_t srand;
	seed_rand(&srand, 1);
	X(randu)(mvm1,1e-7,&srand);
	X(randu)(mvm2,1e-7,&srand);
	X(randu)(mtch, 1, &srand);
	X(randu)(pix1,50, &srand);
	memcpy(pix2->p, pix1->p, sizeof(Real)*totpix);
	X(randu)(ptt, 1, &srand);
	//srandn(pixbias, 1, &srand);
    }
    X(mat) *mvm=mvm1;
    X(mat) *pix=pix2;
    //To receive statistics from GPU
    im0=X(new)(totpix,3);
    if(nstep==1){//Verify accuracy
	//We use half of the array as short.
	writearr("pix", 1, sizeof(short), M_INT16, NULL, pix->p, totpix, 1);
	writearr("pixbias", 1, sizeof(short), M_INT16, NULL, pixbias->p, totpix, 1);
	writebin(mvm1, "mvm1");
	writebin(mtch, "mtch");
    }
    X(cell) *dmres=X(cellnew)(ngpu, 1);
    X(pagelock)(im0, pix1, pix2, mvm1, mvm2, mtch, dmres, NULL);

    int port=20000;
    int sock=-1;
    int ready=1;
    int mtch_ngrid=50;//30;//can change to utilize GPU fully. 16 is good for cassiopeia
    const int mtch_dimx=32;//must launch 32 threads so that they belong to single wrap.
    const int mtch_dimy=12;//number of subapertures
    const int naeach=128;//Each block handle this many subapertures
    int nover=9;//determining how many blocks to launch.
    int nsm=2;//number of streams
    {
	char *MVM_NSM=getenv("MVM_NSM");
	if(MVM_NSM){
	    nsm=strtol(MVM_NSM, NULL, 10);
	    info("nsm is set to %d\n", nsm);
	}
	char *MVM_NGRID=getenv("MVM_NGRID");
	if(MVM_NGRID){
	    mtch_ngrid=strtol(MVM_NGRID, NULL, 10);
	    info("mtch_ngrid is set to %d\n", mtch_ngrid);
	}
	char *MVM_NOVER=getenv("MVM_NOVER");
	if(MVM_NOVER){
	    nover=strtol(MVM_NOVER, NULL, 10);
	}
    }
    const int sastep=mtch_dimy*mtch_ngrid;
    {
	char *MVM_CLIENT=getenv("MVM_CLIENT");
	if(MVM_CLIENT){
	    char *MVM_PORT=getenv("MVM_PORT");
	    if(MVM_PORT){
		port=strtol(MVM_PORT, NULL, 10);
	    }
	    info("Connecting to server %s\n", MVM_CLIENT);
	    sock=connect_port(MVM_CLIENT, port, 0 ,1);
	    if(sock!=-1) {
		info("Connected");
		int cmd[7];
		cmd[0]=nact;
		cmd[1]=nsa;
		cmd[2]=sastep;
		cmd[3]=totpix;
		cmd[4]=nstep;
		cmd[5]=nstep0;
		cmd[6]=2;
		if(stwriteintarr(sock, cmd, 7) 
		   || stwriteintarr(sock, saind, nsa+1)
		   || stwrite(sock, pix->p, sizeof(short)*totpix)){
		    close(sock); sock=-1;
		    warning("Failed: %s\n", strerror(errno));
		}
	    } else {
		info("Failed\n");
	    }
	}
    }

    const int nbuf=2;//two buffers for im0.
    //int dither_nsa=10;//each time step compute this many subapertures for matched filter
    //int comp_mtch_done[ngpu];
    //Real imc, a2m;//PLL results
    int nc=10;//each time copy nc column of mvm.
    GPU_DATA_T **data=new GPU_DATA_T*[ngpu];
    const int sect_gpu=(nsa+sastep*ngpu-1)/(sastep*ngpu);
    for(int igpu=0; igpu<ngpu; igpu++){
	hipSetDevice(gpus[igpu]);
	data[igpu]=new GPU_DATA_T;
	data[igpu]->cumvm1=curmat(nact, ng);
	data[igpu]->cumvm2=curmat(nact, ng);
	data[igpu]->cumvm=data[igpu]->cumvm1;
	data[igpu]->cumvm_next=data[igpu]->cumvm2;
	cp2gpu(data[igpu]->cumvm1, mvm);
	data[igpu]->pix=Array<short,Gpu>(totpix, 1);
	data[igpu]->pixbias=Array<short,Gpu>(totpix, 1);
	cp2gpu(data[igpu]->pixbias(), (short*)pixbias->p(), totpix*sizeof(short), hipMemcpyHostToDevice);
	data[igpu]->mtch=curcell(nbuf, 1, totpix*2,1);
	cp2gpu(data[igpu]->mtch[0], mtch);
	data[igpu]->grad=curmat(ng, 1);
	data[igpu]->act=curmat(nact, 1);
	data[igpu]->actelse=curcell(ngpu-1, 1, nact, 1);
	data[igpu]->im0=curcell(nbuf,1, totpix, 3);//two buffers
	data[igpu]->FSMdelta=0.2;
	data[igpu]->stream_a=Array<stream_t>(nsm, 1);
	data[igpu]->event_w=new event_t[nsm];
	data[igpu]->gpu=gpus[igpu];
	data[igpu]->event_g=new hipEvent_t[sect_gpu];
	data[igpu]->event_p=new hipEvent_t[sect_gpu];
	for(int i=0; i<sect_gpu; i++){
	    hipEventCreateWithFlags(&data[igpu]->event_g[i],event_flag);
	    hipEventCreateWithFlags(&data[igpu]->event_p[i],event_flag);
	}
	hipEventCreateWithFlags(&data[igpu]->event_pall,event_flag);
	dmres->p[igpu]=X(new)(nact, 1);
	X(pagelock)(dmres->p[igpu], NULL);
	data[igpu]->saind=cuimat(nsa+1,1);
	cp2gpu(data[igpu]->saind(), saind, nsa+1, 1, 0);
    }
    X(mat) *timing=X(new)(nstep, 1);
    X(mat) *timing_tot=X(new)(nstep, 1);
    X(mat) *timing_sock=X(new)(nstep, 1);
    hipProfilerStop();
    hipProfilerStart();
    TIC;
    if(sock!=-1 && stwriteint(sock, ready)){
	warning("error send ready signal: %s\n", strerror(errno));
	close(sock); sock=-1;
    }
    int nblock;
    info("Ready\n");
    int ibuf=0;//buffer to use for statistics
    int ibuf_mtch=0;//buffer for matched filter
    int ibuf_stat=0;//buffer for matched filter computation
    int mtch_down=0;
    int nset=(nsa+sastep-1)/sastep;
    char *copied_mtch=(char*)calloc(nset*3, sizeof(char));
    Real tim_tot=0, tim_min=INFINITY, tim_max=0;
    for(int jstep=-nstep0; jstep<nstep; jstep++){
	//run 20 frames to warm up before timing.
	int istep=jstep<0?0:jstep;
	if(sock!=-1){//start signal
	    timing_sock->p[istep]=0;
	}
	tic;
	if(nover>0){
	    nblock=(nact*nover+naeach-1)/naeach;
	}else{
	    nblock=(nact*(1+istep/50)+naeach-1)/naeach;
	}
	if((1+istep)%8000==0){//need to update MVM
	    if(mvm==mvm1){//switch mvm on host.
		mvm=mvm2;
	    }else{
		mvm=mvm1;
	    }
	    for(int igpu=0; igpu<ngpu; igpu++){
		data[igpu]->copy_mvm=1;
		if(data[igpu]->ic!=0){
		    warning("Sync error, skip update request at step %d\n", istep);
		}
	    }
	}
	for(int igpu=0; igpu<ngpu; igpu++){
	    data[igpu]->ism=-1;
	    data[igpu]->count=0;
	    data[igpu]->istep=istep;
	}
	if(sock==-1){
	    if(pix==pix1){
		pix=pix2;
	    }else{
		pix=pix1;
	    }
	}

	for(int isa=0, igpu=0; isa<nsa; isa+=sastep, igpu=((igpu+1)%ngpu)){
	    hipSetDevice(gpus[igpu]); 
	    GPU_DATA_T *datai=data[igpu];
	    int npixleft;
	    int nsaleft;
	    if(nsa<isa+sastep){//terminate
		npixleft=totpix-saind[isa];
		nsaleft=nsa-isa;
	    }else{
		npixleft=saind[isa+sastep]-saind[isa];
		nsaleft=sastep;
	    }
	    //One stream handling the memcpy
	    short *pcur=(short*)(pix->p())+saind[isa];
	    if(sock!=-1){
		double tmp0=myclockd();
		if(stread(sock, pcur, sizeof(short)*npixleft)){
		    warning("failed: %s\n", strerror(errno));
		    close(sock); sock=-1;
		    _Exit(1);
		}
		timing_sock->p[istep]+=myclockd()-tmp0;
	    }
	 
	    DO(hipMemcpyAsync(datai->pix()+saind[isa], pcur, sizeof(short)*npixleft,
			       hipMemcpyHostToDevice, datai->stream_p));
	    //Recored the event when the memcpy is finished
	    DO(hipEventRecord(datai->event_p[datai->count], datai->stream_p));
	    //Start matched filter when pixel transfer is done.
	    DO(hipStreamWaitEvent(datai->stream_g, datai->event_p[datai->count], 0));

	    mtch_do<<<mtch_ngrid, dim3(mtch_dimx, mtch_dimy), 
		mtch_dimx*mtch_dimy*sizeof(Real)*2, datai->stream_g>>>
		(datai->mtch[ibuf_mtch], datai->pix, datai->pixbias, 
		 datai->grad()+isa*2, datai->saind()+isa, nsaleft);
	    //Record the event when matched filter is done
	    DO(hipEventRecord(datai->event_g[datai->count], datai->stream_g));
	    //Another stream does the matrix vector multiplication. Wait for the event before executing.
	    //The stream stream will wait only for the completion of the most recent host call to hipEventRecord() on event
	    datai->ism=(datai->ism+1)%nsm;
	    /*
	      Todo: 
	      *) Project PTT modes. Generate LLT FSM signal.
	      *) Minus LPF focus mode
	      *) Add gradient from DM to form PSOL grads.
	    */
	    hipStreamWaitEvent(datai->stream_a[datai->ism], datai->event_g[datai->count], 0);

#if 0
	    Real one=1;
	    DO(CUBL(gemv)(datai->stream_a[datai->ism], HIPBLAS_OP_N, nact, nsaleft*2, &one, datai->cumvm->p+nact*isa*2, nact, datai->grad->p+isa*2, 1, &one, datai->act->p, 1));
#else
	    multimv_do<<<nblock, naeach, sizeof(Real)*naeach, datai->stream_a[datai->ism]>>>
		(datai->cumvm()+nact*isa*2, datai->act, datai->grad()+isa*2, 
		 nact, nsaleft*2);
#endif
	    DO(hipEventRecord(datai->event_w[datai->ism], datai->stream_a[datai->ism]));
	    datai->count++;
	}
	for(int igpu=0; igpu<ngpu; igpu++){
	    GPU_DATA_T *datai=data[igpu];
	    //Record an event when pixel tranporting is over. So we can start transporting mvm matrix.
	    DO(hipEventRecord(datai->event_pall, datai->stream_p));
	}
	/*Accumulate statistics in each cycle. The pixels are present in different GPUs*/
	for(int isa=0, igpu=0; isa<nsa; isa+=sastep, igpu=((igpu+1)%ngpu)){
	    hipSetDevice(gpus[igpu]); 
	    GPU_DATA_T *datai=data[igpu];
	    int npixleft;
	    if(nsa<isa+sastep){//terminate
		npixleft=totpix-saind[isa];
	    }else{
		npixleft=saind[isa+sastep]-saind[isa];
	    }
	    double theta=M_PI*0.5*istep+datai->FSMdelta;
	    Real cd=cos(theta);
	    Real sd=cos(theta);
	    //Do not start before pixels are transported
#if 0
	    for(int ism=1; ism<nsm; ism++){//wait for MVM
		DO(hipStreamWaitEvent(datai->stream_b, datai->event_w[ism], 0));
	    }
#else
	    //Wait for pixel transfer
	    hipStreamWaitEvent(datai->stream_b, datai->event_pall, 0);
#endif
	    dither_acc_do<<<DIM(npixleft, 256), 0, datai->stream_b>>>
		(datai->pix()+saind[isa], datai->im0[ibuf]()+saind[isa], 
		 datai->im0[ibuf]()+totpix+saind[isa],datai->im0[ibuf]()+totpix*2+saind[isa],
		 cd, sd, npixleft);
	}
	//Download statistics to CPU for matched filter building.
	if(mtch_down || nstep==1){
	    hipStream_t stream;
	    int iset=0;
	    for(int icol=0; icol<3; icol++){
		for(int isa=0, igpu=0; isa<nsa; isa+=sastep, igpu=((igpu+1)%ngpu), iset++){
		    if(copied_mtch[iset]) continue;
		    hipSetDevice(gpus[igpu]); 
		    if(nstep==1){
			stream=data[igpu]->stream_b;
		    }else{
			stream=data[igpu]->stream_p;
		    }
		    int npixleft;
		    if(nsa<isa+sastep){//terminate
			npixleft=totpix-saind[isa];
		    }else{
			npixleft=saind[isa+sastep]-saind[isa];
		    }
		    DO(hipMemcpyAsync(im0->p+saind[isa]+icol*totpix, 
				       data[igpu]->im0[ibuf_stat]()+saind[isa]+icol*totpix,
				       sizeof(Real)*npixleft,
				       hipMemcpyDeviceToHost, stream));
		    copied_mtch[iset]=1;
		    if(nstep!=1) goto endhere;
		}
	    }
	    mtch_down=0;//completed
	  endhere:;
	}
	//Queue copying MVM matrix to second slot.
	for(int igpu=0; igpu<ngpu; igpu++){
	    GPU_DATA_T *datai=data[igpu];
	    if(datai->copy_mvm){
		int done=0, nsaleft;
		if(mvm->ny-datai->ic < nc){
		    done=1;
		    nsaleft=mvm->ny-datai->ic;
		}else{
		    nsaleft=nc;
		}
		if(datai->ic==0){
		    info("step %d: gpu %d uploading mvm\n", istep, igpu);
		}
		DO(hipMemcpyAsync(datai->cumvm_next()+datai->ic*mvm->nx, 
				   mvm->p+datai->ic*mvm->nx, sizeof(Real)*mvm->nx*nsaleft, 
				   hipMemcpyHostToDevice, datai->stream_p));
		
		datai->ic+=nsaleft;
		if(done){
		    datai->ic=0;
		    datai->copy_mvm=0;
		    curmat tmp=datai->cumvm;
		    datai->cumvm=datai->cumvm_next;
		    datai->cumvm_next=tmp;
		    info("gpu %d switched over at step %d\n", datai->gpu, datai->istep);
		}
	    }
	}
#if 1	//CPU sum
	//Copy DM commands back to CPU
	for(int igpu=0; igpu<ngpu; igpu++){
	    GPU_DATA_T *datai=data[igpu];
	    hipSetDevice(gpus[igpu]); 
	    for(int ism=1; ism<nsm; ism++){
		DO(hipStreamWaitEvent(datai->stream_a[0], datai->event_w[ism], 0));
	    }
	    hipMemcpyAsync(dmres->p[igpu]->p, datai->act, nact*sizeof(Real), 
			    hipMemcpyDeviceToHost, datai->stream_a[0]);
	    Zero(datai->act, datai->stream_a[0]);
	}
	//CPU sums them together. sync first gpu
	data[0]->stream_a[0].sync();
	//sum other GPUs
	for(int igpu=1; igpu<ngpu; igpu++){
	    hipSetDevice(gpus[igpu]); 
	    data[igpu]->stream_a[0].sync();
	    for(int iact=0; iact<nact; iact++){
		dmres->p[0]->p[iact]+=dmres->p[igpu]->p[iact];
	    }
	}
#else //GPU sum
	//First copy second GPU to first GPU.
	for(int igpu=1; igpu<ngpu; igpu++){
	    GPU_DATA_T *datai=data[igpu];
	    hipSetDevice(gpus[igpu]); 
	    for(int ism=1; ism<nsm; ism++){
		DO(hipStreamWaitEvent(datai->stream_a[0], datai->event_w[ism], 0));
	    }
	    hipMemcpyAsync(data[0].actelse->p[igpu-1]->p, datai->act->p, nact*sizeof(Real), 
			    hipMemcpyDeviceToDevice, datai->stream_a[0]);
	}
	if(ngpu>1){
	    if(ngpu==2){
		int igpu=0;
		hipSetDevice(gpus[igpu]);
		GPU_DATA_T *datai=data[igpu];
		for(int ism=1; ism<nsm; ism++){
		    DO(hipStreamWaitEvent(datai->stream_a[0], datai->event_w[ism], 0));
		}
		add_do<<<DIM(nact, 256), 0, datai->stream_a[0]>>>
		    (datai->act->p, datai->actelse->p[0]->p, (Real*)0, 1, nact);
		hipMemcpyAsync(dmres->p[0]->p, datai->act->p, nact*sizeof(Real), 
				hipMemcpyDeviceToHost, datai->stream_a[0]);
		datai->stream_a[0].sync();
	    }else{
		error("Please implement\n");
	    }
	}
#endif
	/*
	  Save resutls for debugging.
	 */
	//usleep(50);//yield
	if(nstep==1){//save result for verifying accuracy
	    writebin(dmres->p[0], "dmres");
	    for(int igpu=0; igpu<ngpu; igpu++){
		hipSetDevice(gpus[igpu]); 
		hipMemcpy(pix->p, data[igpu]->pix, sizeof(short)*totpix, hipMemcpyDefault);
		char fn[PATH_MAX];
		snprintf(fn, PATH_MAX, "pix_gpu%d", igpu);
		writearr(fn, 1, sizeof(short), M_INT16, NULL, pix->p, totpix, 1);
		cuwrite(data[igpu]->grad, "grad_gpu%d", igpu);
	    }
	}
	/*
	  ToDO: the following background process.
	  *) Update PLL each time step
	  *) Output PLL results every 240 steps
	 */

	if(istep>0 && (istep+1)%2400==0){
	    ibuf_stat=ibuf;
	    info("Download statistics to CPU at step %d\n", istep);
	    ibuf=(ibuf+1)%nbuf;
	    mtch_down=1;
	    memset(copied_mtch, 0, sizeof(char)*nset*3);
	}
	for(int igpu=0; igpu<ngpu; igpu++){
	    GPU_DATA_T *datai=data[igpu];
	    hipSetDevice(datai->gpu);
	    //no need to zero gradients.
	    datai->stream_b.sync();
	    datai->stream_p.sync();
	}
	if(sock!=-1){
	    double tmp0=myclockd();
	    if(stwrite(sock, dmres->p[0]->p, sizeof(Real)*nact)){
		warning("error write dmres: %s\n", strerror(errno));
		close(sock); sock=-1;
		_Exit(1);
	    }
	    if(streadint(sock, &ready)){//acknowledgement.
		warning("error read ack failed: %s\n", strerror(errno));
		close(sock), sock=-1;
		_Exit(1);
	    }
	    timing_sock->p[istep]+=myclockd()-tmp0;
	    timing->p[istep]=ready*1.e-6;
	}else{
	    timing->p[istep]=toc3;//do not tic.
	}
	timing_tot->p[istep]=toc3;
	if(istep>0){
	    tim_tot+=timing->p[istep];
	    if(tim_min>timing->p[istep]) tim_min=timing->p[istep];
	    if(tim_max<timing->p[istep]) tim_max=timing->p[istep];
	    if(istep%1000==0){
		info("Step %d, mean=%g, min=%g, max=%g ms\n", istep, tim_tot/(istep)*1e3, tim_min*1e3, tim_max*1e3);
	    }
	}
    }
    hipProfilerStop();
    //writebin(dmres->p[0], "dmres");
    
    writebin(timing, "timing_%s_%dgpu", HOST, ngpu);
    writebin(timing_tot, "timing_tot_%s_%dgpu", HOST, ngpu);
    writebin(timing_sock, "timing_sock_%s_%dgpu", HOST, ngpu);
    X(pageunlock)(pix1, pix2, mvm1, mvm2, NULL);
    X(free)(mvm1);
    X(free)(mvm2);
    X(free)(pix1);
    X(free)(pix2);
    X(free)(mtch);
    X(cellfree)(dmres);
    X(free)(timing);
    X(free)(timing_tot);
    X(free)(timing_sock);
    for(int igpu=0; igpu<ngpu; igpu++){
	hipSetDevice(gpus[igpu]);
	delete[] data[igpu]->event_w;
	delete[] data[igpu]->event_g;
	delete[] data[igpu]->event_p;
	hipDeviceReset();
    }
    free(data);
  
}
