/*
  Copyright 2009-2019 Lianqi Wang <lianqiw-at-tmt-dot-org>
  
  This file is part of Multithreaded Adaptive Optics Simulator (MAOS).

  MAOS is free software: you can redistribute it and/or modify it under the
  terms of the GNU General Public License as published by the Free Software
  Foundation, either version 3 of the License, or (at your option) any later
  version.

  MAOS is distributed in the hope that it will be useful, but WITHOUT ANY
  WARRANTY; without even the implied warranty of MERCHANTABILITY or FITNESS FOR
  A PARTICULAR PURPOSE.  See the GNU General Public License for more details.

  You should have received a copy of the GNU General Public License along with
  MAOS.  If not, see <http://www.gnu.org/licenses/>.
*/
#include "utils.h"
#include "accphi.h"
#include "cucmat.h"
#include "kernel.h"
#include "cudata.h"
#include "perf.h"

/**
   Initialize perfevl
*/
void gpu_perfevl_init(const PARMS_T *parms, APER_T *aper){
    if(!parms->gpu.evl){
	return;
    }
    const int nevl=parms->evl.nevl;
    const int nwvl=parms->evl.nwvl;
    /*The following lives in CPU. */
    if(parms->evl.psfmean || parms->evl.psfhist){
	cuglobal->perf.nembed =Array<int>(nwvl, 1);
	cuglobal->perf.psfsize=Array<int>(nwvl, 1);
	cuglobal->perf.wvls   =Array<Real>(nwvl, 1);
    
	for(int iwvl=0; iwvl<nwvl; iwvl++){
	    cuglobal->perf.nembed[iwvl]=(int)aper->embed->nembed->p[iwvl];
	    cuglobal->perf.psfsize[iwvl]=parms->evl.psfsize->p[iwvl];
	    cuglobal->perf.wvls[iwvl]=parms->evl.wvl->p[iwvl];
	}
    }
    /*The following lives in GPU. */
    for(int im=0; im<NGPU; im++){
	gpu_set(im);
	cudata->perf.locs=culoc_t(aper->locs);
	cp2gpu(cudata->perf.amp, aper->amp);
	cp2gpu(cudata->perf.imcc, aper->imcc);
	if(parms->evl.psfmean || parms->evl.psfhist){
	    cudata->perf.embed    = Array<cuimat>(nwvl, 1);//(int**) calloc(nwvl, sizeof(int*));
	    for(int iwvl=0; iwvl<nwvl; iwvl++){
		cp2gpu(cudata->perf.embed[iwvl], aper->embed->embed->p[iwvl]->p(), aper->locs->nloc, 1);
	    }
	}
    }/*for igpu */
    for(int ievl=0; ievl<nevl; ievl++){
	gpu_set(cuglobal->evlgpu[ievl]);
	if(!cudata->perf.locs_dm){
	    cudata->perf.locs_dm=Array<Array<culoc_t> >(nevl, 1);
	}
	cudata->perf.locs_dm[ievl]=Array<culoc_t>(parms->ndm,1);
	for(int idm=0; idm<parms->ndm; idm++){
	    loc_t *loc_dm;
	    if(aper->locs_dm && aper->locs_dm->p[ievl+idm*nevl]){
		loc_dm=aper->locs_dm->p[ievl+idm*nevl];
	    }else{
		loc_dm=aper->locs;
	    }
	    cudata->perf.locs_dm[ievl][idm]=culoc_t(loc_dm);
	}
    }
    cuglobal->perf.stream=Array<stream_t>(nevl, 1);
    if(parms->evl.psfmean || parms->evl.psfhist){
	cuglobal->perf.plan  = Array<hipfftHandle>(nwvl*nevl,1);
    }
    for(int ievl=0; ievl<nevl; ievl++){
	gpu_set(cuglobal->evlgpu[ievl]);
	//STREAM_NEW(cuglobal->perf.stream[ievl]);
	//Use stream created per GPU in order to share resource within GPU between different evl dir.
	cuglobal->perf.stream[ievl]=cudata->perf_stream;
	if(parms->evl.psfmean || parms->evl.psfhist){
	    for(int iwvl=0; iwvl<nwvl; iwvl++){
		DO(hipfftPlan2d(&cuglobal->perf.plan[iwvl+nwvl*ievl],cuglobal->perf.nembed[iwvl],
			       cuglobal->perf.nembed[iwvl],FFT_T_C2C));
		DO(hipfftSetStream(cuglobal->perf.plan[iwvl+nwvl*ievl], cuglobal->perf.stream[ievl]));
	    }/*for iwvl */
	}
    }
    cuglobal->perf.nevl=nevl;
    cuglobal->perf.opd=curcell(nevl,1);
    cuglobal->perf.cc_cl=curcell(nevl, 1);
    cuglobal->perf.cc_ol=curcell(nevl, 1);
    cuglobal->perf.coeff=curcell(nevl, 1);
    cuglobal->perf.ccb_ol.init(nevl,1);
    cuglobal->perf.ccb_cl.init(nevl,1);
    //cuglobal->perf.ccb_ol=(Real**)malloc(sizeof(Real*)*nevl);
    //cuglobal->perf.ccb_cl=(Real**)malloc(sizeof(Real*)*nevl);
    for(int ievl=0; ievl<nevl; ievl++){
	gpu_set(cuglobal->evlgpu[ievl]);
	cuglobal->perf.ccb_ol[ievl].init(7,1);//=(Real*)malloc4async(sizeof(Real)*7);
	cuglobal->perf.ccb_cl[ievl].init(7,1);//=(Real*)malloc4async(sizeof(Real)*7);
	cuglobal->perf.cc_cl[ievl].init(7,1);
	cuglobal->perf.cc_ol[ievl].init(7,1);
	cuglobal->perf.coeff[ievl].init(7,1);
	cuglobal->perf.opd[ievl].init(aper->locs->nloc, 1);
    }
    if(!parms->sim.evlol){
	if(parms->evl.cov && parms->gpu.psf){
	    cuglobal->perf.opdcov=curcell(nevl, 1);
	    cuglobal->perf.opdmean=curcell(nevl, 1);
	    cuglobal->perf.opdcov_ngsr=curcell(nevl, 1);
	    cuglobal->perf.opdmean_ngsr=curcell(nevl, 1);
	}
	if(parms->evl.psfmean || parms->evl.psfhist){
	    cuglobal->perf.psfcl = curcell(nwvl, parms->evl.nevl);
	    cuglobal->perf.psfcl_ngsr = curcell(nwvl, parms->evl.nevl);
	}
    }
    if(aper->opdadd){
	cuglobal->perf.surf=curcell(nevl, 1);
	for(int ievl=0; ievl<nevl; ievl++){
	    gpu_set(cuglobal->evlgpu[ievl]);
	    cp2gpu(cuglobal->perf.surf[ievl], aper->opdadd->p[ievl]);
	}
    }
    gpu_print_mem("perf init");
}
/*
  Initialize simulation data. Seed dependent. Create for the first seed and zero for the next.
*/
void gpu_perfevl_init_sim(const PARMS_T *parms, APER_T *aper){
    const int nevl=parms->evl.nevl;
    const int nwvl=parms->evl.nwvl;
    int nloc=aper->locs->nloc;
    if(!parms->gpu.evl){
	return;
    }
    /*first open loop ones are on every GPU.*/
    if(parms->evl.psfol){
	for(int im=0; im<NGPU; im++){
	    gpu_set(im);
	    if(parms->evl.cov && parms->gpu.psf){ /*do OL opd cov*/
		cudata->perf.opdcovol.init(nloc, nloc);
		cudata->perf.opdmeanol.init(nloc, 1);
	    }
	    if(parms->evl.psfmean || parms->evl.psfhist){
		if(cudata->perf.psfol){
		    Zero(cudata->perf.psfol);
		}else{
		    cudata->perf.psfol=curcell(nwvl,1);
		    for(int iwvl=0; iwvl<nwvl; iwvl++){
			cudata->perf.psfol[iwvl]=curmat(cuglobal->perf.psfsize[iwvl], 
							cuglobal->perf.psfsize[iwvl]);
		    }
		}
	    }
	}
    }

    if(parms->evl.cov && parms->gpu.psf && !parms->sim.evlol){
	for(int ievl=0; ievl<nevl; ievl++){
	    if(parms->evl.psf->p[ievl]==0){
		continue;
	    }
	    gpu_set(cuglobal->evlgpu[ievl]);
	    if(parms->evl.psfngsr->p[ievl]){
		cuglobal->perf.opdcov_ngsr[ievl].init(nloc,nloc);
		cuglobal->perf.opdmean_ngsr[ievl].init(nloc,1);
	    }
	    if(parms->evl.psfngsr->p[ievl]!=2){
		cuglobal->perf.opdcov[ievl].init(nloc,nloc);
		cuglobal->perf.opdmean[ievl].init(nloc,1);
	    }
	}
    }
	
    if(parms->evl.psfmean || parms->evl.psfhist){
	if(!parms->sim.evlol){
	    for(int ievl=0; ievl<nevl; ievl++){
		if(parms->evl.psf->p[ievl]==0){
		    continue;
		}
		gpu_set(cuglobal->evlgpu[ievl]);
		for(int iwvl=0; iwvl<nwvl; iwvl++){
		    if(parms->evl.psfngsr->p[ievl]){
			cuglobal->perf.psfcl_ngsr[iwvl+nwvl*ievl].init(
			    cuglobal->perf.psfsize[iwvl], cuglobal->perf.psfsize[iwvl]);
		    }
		    if(parms->evl.psfngsr->p[ievl]!=2){
			cuglobal->perf.psfcl[iwvl+nwvl*ievl].init(
			    cuglobal->perf.psfsize[iwvl], cuglobal->perf.psfsize[iwvl]);
		    }
		}	
	    }
	}
    }
}
