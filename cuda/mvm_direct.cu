/*
  Copyright 2009-2019 Lianqi Wang <lianqiw-at-tmt-dot-org>
  
  This file is part of Multithreaded Adaptive Optics Simulator (MAOS).

  MAOS is free software: you can redistribute it and/or modify it under the
  terms of the GNU General Public License as published by the Free Software
  Foundation, either version 3 of the License, or (at your option) any later
  version.

  MAOS is distributed in the hope that it will be useful, but WITHOUT ANY
  WARRANTY; without even the implied warranty of MERCHANTABILITY or FITNESS FOR
  A PARTICULAR PURPOSE.  See the GNU General Public License for more details.

  You should have received a copy of the GNU General Public License along with
  MAOS.  If not, see <http://www.gnu.org/licenses/>.
*/
#include "utils.h"
#include "wfs.h"
#include "recon.h"
#include "pcg.h"
#include "curmat.h"
#include "cucmat.h"
#include "accphi.h"
#include "cudata.h"
/**
   \file mvm_direct.cu
   Compute the MVM control matrix in GPU colume by colume
*/
static int *gpu_avail=NULL;//implementes a single stack to manage available GPUs
static int gpu_pos=0;
PNEW(gpu_mutex);
typedef struct{
    const PARMS_T *parms;
    const RECON_T *recon;
    dmat *residual;
    dmat *residualfit;
    long ntotact;
    long ntotgrad;
    long ntotxloc;
    X(mat) *mvmc;
    X(mat) *mvmi;
}MVM_IGPU_T;
#define TIMING 0
static void mvm_direct_igpu(thread_t *info){
    int igpu=info->ithread;
    if(gpu_avail){
	LOCK(gpu_mutex);
	if(gpu_pos>0){
	    igpu=gpu_avail[--gpu_pos];
	}else{
	    igpu=-1;
	    warning("error usage\n");
	}
	UNLOCK(gpu_mutex);
    }
    if(igpu==-1) return;
    gpu_set(igpu);
    info("thread %ld is using GPU %d\n", info->ithread, igpu);
#if TIMING
#define RECORD(i) DO(hipEventRecord(event[i], stream))
#define NEVENT 4
    hipEvent_t event[NEVENT]={0};
    Real times[NEVENT];
    for(int i=0; i<NEVENT; i++){
	DO(hipEventCreate(&event[i]));
    }
#else
#define RECORD(i)
#endif

    MVM_IGPU_T *data=(MVM_IGPU_T*)info->data;
    const PARMS_T *parms=data->parms;
    const RECON_T *recon=data->recon;
    const long ntotact=data->ntotact;
    const long ntotgrad=data->ntotgrad;
    const long ntotxloc=data->ntotxloc;
    curcell grad=curcell(parms->nwfsr, 1, recon->ngrad->p, (long*)NULL);//the I
    curcell opdx=curcell(recon->npsr, 1, recon->xnx->p, recon->xny->p);//right hand size
    curcell opdr;//initialized later
    //curcell *fitx=curcellnew(parms->ndm, 1, recon->anloc->p, (long*)NULL);
    curcell fitr=curcell(parms->ndm, 1, recon->anloc->p, (long*)NULL, (Real*)1);//skip data allocation.
    curmat mvm=curmat(ntotact, info->end-info->start);
    curmat eye2(2,1);
    dmat *residual=data->residual;
    dmat *residualfit=data->residualfit;
    {
	Real eye2c[2]={0,1.};
	hipMemcpy(eye2(), eye2c, sizeof(Real)*2, hipMemcpyHostToDevice);
    }
    cuda_recon::curecon_t *curecon=cudata->recon;
    stream_t stream;
    if(parms->load.mvmf){
	hipMemcpyAsync(mvm(), data->mvmc->p+info->start*ntotact, 
			ntotact*(info->end-info->start)*sizeof(Real), 
			hipMemcpyHostToDevice, stream);
    }
    curmat mvmi;
    if(parms->load.mvmi || parms->save.mvmi){
	dbg("Creating mvmi of size %ldx %ld\n", ntotxloc, info->end-info->start);
	mvmi=curmat(ntotxloc, info->end-info->start);
	if(parms->load.mvmi){
	    hipMemcpyAsync(mvmi(), data->mvmi->p+info->start*ntotxloc,
			    ntotxloc*(info->end-info->start)*sizeof(Real),
			    hipMemcpyHostToDevice, stream);
	}
	opdr=curcell(recon->npsr, 1, recon->xnx->p, recon->xny->p, (Real*)1);
    }else{
	opdr=curcell(recon->npsr, 1, recon->xnx->p, recon->xny->p);
    }
    TIC;tic;
    curcell tomo_rhs, fit_rhs;
    for(int ig=info->start; ig<info->end; ig++){
	RECORD(0);
	if(info->ithread==0){
	    if(!detached){
		info2("%6d of %6ld\b\b\b\b\b\b\b\b\b\b\b\b\b\b\b\b", ig*NGPU, ntotgrad);
	    }else if(ig % 100==0){
		info2("%6d of %6ld\n", ig*NGPU, ntotgrad);
	    }
	}
	if(ig){
	    hipMemcpyAsync(grad.M()()+ig-1, eye2(), 2*sizeof(Real), hipMemcpyDeviceToDevice, stream);
	}else{
	    hipMemcpyAsync(grad.M()()+ig, eye2()+1, sizeof(Real), hipMemcpyDeviceToDevice, stream);
	}
	RECORD(1);
	if(mvmi){
	    opdr.Replace(mvmi()+(ig-info->start)*ntotxloc, stream);
	}
	curecon->RR->R(tomo_rhs, 0, grad, 1, stream);
	residual->p[ig]=curecon->RL->solve(opdr, tomo_rhs, stream);
	RECORD(2);
	fitr.Replace(mvm()+(ig-info->start)*ntotact, stream);
	curecon->FR->R(fit_rhs, 0, opdr, 1, stream);
	residualfit->p[ig]=curecon->FL->solve(fitr, fit_rhs, stream);
	RECORD(3);
#if TIMING
	stream.sync();
	for(int i=1; i<NEVENT; i++){
	    DO(hipEventElapsedTime(&times[i], event[i-1], event[i]));
	    times[i]*=1e3;//micro-second
	}
	info("copy=%3.0f, Tomo=%3.0f, Fit=%3.0f\n", times[1], times[2], times[3]);
#endif	
    }
    DO(hipMemcpyAsync(data->mvmc->p+info->start*ntotact, 
		       mvm(), ntotact*(info->end-info->start)*sizeof(Real), 
		       hipMemcpyDeviceToHost, stream));
    if(parms->save.mvmi){
	DO(hipMemcpyAsync(data->mvmi->p+info->start*ntotxloc,
			   mvmi(), ntotxloc*(info->end-info->start)*sizeof(Real),
			   hipMemcpyDeviceToHost, stream));
    }
    stream.sync();
    toc2("Thread %ld mvm", info->ithread);
    if(gpu_avail){
	LOCK(gpu_mutex);
	gpu_avail[gpu_pos++]=igpu;
	UNLOCK(gpu_mutex);
    }
    dbg("thread %ld finish.\n", info->ithread);
}
/**
   Assemble the MVM control matrix.
*/
void gpu_setup_recon_mvm_direct(const PARMS_T *parms, RECON_T *recon){
    TIC;tic;
    if(parms->recon.alg!=0){
	error("Please adapt to LSR\n");
    } 
    if(!parms->load.mvm){
	info("Assembling MVR MVM (direct) in GPU\n");
	
	long ntotact=0;
	long ntotgrad=0;
	long ntotxloc=0;
	const int ndm=parms->ndm;
	for(int idm=0; idm<ndm; idm++){
	    ntotact+=recon->anloc->p[idm];
	} 
	for(int ips=0; ips<recon->npsr; ips++){
	    ntotxloc+=recon->xloc->p[ips]->nloc;
	}
	for(int iwfs=0; iwfs<parms->nwfsr; iwfs++){
	    ntotgrad+=recon->ngrad->p[iwfs];
	}
	X(mat) *mvmc=NULL;//control matrix output to CPU
	if(parms->load.mvmf){
	    mvmc=X(read)("%s", parms->load.mvmf);
	}else{
	    mvmc=X(new)(ntotact, ntotgrad);
	}
	X(mat) *mvmi=NULL;//intermediate result
	if(parms->load.mvmi){
	    tic; info("Loading mvmi ...");
	    mvmi=X(read)("%s", parms->load.mvmi);
	    toc2("done");
	}else if(parms->save.mvmi){
	    mvmi=X(new)(ntotxloc, ntotgrad);
	}
	dmat *residual=dnew(ntotgrad,1);
	dmat *residualfit=dnew(ntotgrad, 1);
	MVM_IGPU_T data={parms, recon, residual, residualfit, ntotact, ntotgrad, ntotxloc, mvmc, mvmi};
	int nthread=NGPU;
	if(parms->load.mvmi || parms->save.mvmi){
	    /*Each GPU cannot handle all the mvmi if just divide to NGPU
	      runs. Do multiple pass to avoid memroy overflow. Assemes each GPU
	      has more than 2G free space.*/
	    int npass=iceil((double)ntotxloc*(double)ntotgrad*sizeof(Real)/NGPU/2000000000);
	    dbg("mul=%ld\n", ntotxloc*ntotgrad*sizeof(Real));
	    dbg("NGPU=%d\n", NGPU);
	    dbg("npass=%d\n", npass);
	    nthread=NGPU*npass;
	}
	thread_t info[nthread];
	thread_prep(info, 0, ntotgrad, nthread, mvm_direct_igpu, &data);
	if(nthread>NGPU){
	    THREAD_POOL_INIT(NGPU);//limit to only NGPU threads to avoid fighting
	    sleep(1);
	    gpu_avail=(int*)calloc(NGPU, sizeof(int));
	    for(int igpu=0; igpu<NGPU; igpu++){
		gpu_avail[gpu_pos++]=igpu;
	    }
	}
	CALL_THREAD(info, 1);
	if(nthread>NGPU){
	    THREAD_POOL_INIT(NTHREAD);
	    free(gpu_avail); gpu_avail=NULL;
	}
	toc("Assembly");tic;
	dfree(residual);
	dfree(residualfit);
	if(parms->save.mvmf){
	    writebin(mvmc, "mvmf.bin");
	}
	if(parms->save.mvmi){
	    tic; info("Saving mvmi ...");
	    writebin(mvmi, "mvmi.bin");
	    toc2("done");
	}
	X(free)(mvmi);
	{
	    dmat *dmvmc=dnew(ntotact, ntotgrad);
	    for(long i=0; i<ntotact*ntotgrad; i++){
		dmvmc->p[i]=(double)mvmc->p[i];
	    }
	    recon->MVM=dmvmc;
	    X(free)(mvmc);
	}
    }
}

